/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <ParTI.h>

int main(int argc, char const *argv[]) {
    FILE *fA, *fB, *fY;
    sptSparseTensor A, B, Y;
    int cuda_dev_id = -2;

    if(argc < 4) {
        printf("Usage: %s A B Y [cuda_dev_id]\n\n", argv[0]);
        exit(1);
    }

    fA = fopen(argv[1], "r");
    if(!fA) {
        fprintf(stderr, "Error: failed to open file \"%s\"\n", argv[1]);
        exit(1);
    }
    if(sptLoadSparseTensor(&A, 1, fA)) {
        fprintf(stderr, "Error: failed to load tensor A\n");
        exit(1);
    }
    fclose(fA);

    fB = fopen(argv[2], "r");
    if(!fB) {
        fprintf(stderr, "Error: failed to open file \"%s\"\n", argv[2]);
        exit(1);
    }
    if(sptLoadSparseTensor(&B, 1, fB)) {
        fprintf(stderr, "Error: failed to load tensor B\n");
        exit(1);
    }
    fclose(fB);

    if(argc >= 6) {
        sscanf(argv[5], "%d", &cuda_dev_id);
    }

    int result;
    if(cuda_dev_id == -2) {
        result = sptSparseTensorDotMulEq(&Y, &A, &B);
    } else if(cuda_dev_id == -1) {
        result = sptOmpSparseTensorDotMulEq(&Y, &A, &B);
    } else {
        result = sptCudaSparseTensorDotMulEq(&Y, &A, &B);
    }
    if(result) {
        fprintf(stderr, "Error: failed to calculate A.*B\n");
        exit(1);
    }

    sptFreeSparseTensor(&B);
    sptFreeSparseTensor(&A);

    fY = fopen(argv[3], "w");
    if(!fY) {
        fprintf(stderr, "Error: failed to open file \"%s\"\n", argv[3]);
        exit(1);
    }
    if(sptDumpSparseTensor(&Y, 1, fY)) {
        fprintf(stderr, "Error: failed to dump tensor Y\n");
        exit(1);
    }
    fclose(fY);

    sptFreeSparseTensor(&Y);

    return 0;
}
