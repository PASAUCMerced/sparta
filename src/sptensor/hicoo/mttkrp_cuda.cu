/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include "hicoo.h"
#include "../../cudawrap.h"
#include "mttkrp_cuda_kernels.h"



/**
 * CUDA parallelized Matriced sparse tensor times a sequence of dense matrix Khatri-Rao products (MTTKRP) on a specified mode
 * @param[out] mats[nmodes]    the result of MTTKRP, a dense matrix, with size
 * ndims[mode] * R
 * @param[in]  X    the sparse tensor input X
 * @param[in]  mats    (N+1) dense matrices, with mats[nmodes] as temporary
 * @param[in]  mats_order    the order of the Khatri-Rao products
 * @param[in]  mode   the mode on which the MTTKRP is performed
 * @param[in]  scratch an temporary array to store intermediate results, space assigned before this function
 *
 * This function uses support arbitrary-order sparse tensors with Khatri-Rao
 * products of dense factor matrices, the output is the updated dense matrix for the "mode".
 * In this version, atomic function to lock the global reduction and a large
 * scratch is used to maximize parallelism. (To be optimized)
 */
int sptCudaMTTKRPHiCOO(
    sptSparseTensorHiCOO const * const hitsr,
    sptMatrix ** const mats,     // mats[nmodes] as temporary space.
    sptIndex * const mats_order,    // Correspond to the mode order of X.
    sptIndex const mode,
    sptNnzIndex const max_nnzb,
    int const impl_num)
{
    sptIndex const nmodes = hitsr->nmodes;
    sptNnzIndex const nnz = hitsr->nnz;
    sptIndex const * const ndims = hitsr->ndims;
    sptIndex const R = mats[mode]->ncols;
    sptIndex const stride = mats[mode]->stride;
    int result;

    double time_h2d, time_exe, time_d2h;
    double gbw_h2d, gflops_exe, gbw_d2h;
    sptTimer timer;
    sptNewTimer(&timer, 0);

    /* Check the mats. */
    for(sptIndex i=0; i<nmodes; ++i) {
        if(mats[i]->ncols != mats[nmodes]->ncols) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->cols != mats[nmodes]->ncols");
        }
        if(mats[i]->nrows != ndims[i]) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns MTTKRP", "mats[i]->nrows != ndims[i]");
        }
    }


    /* Copy tensor */
    /* dev_ndims: 1st gpu. */
    sptIndex * dev_ndims;
    /* dev_cptr: 1st gpu. */
    sptNnzIndex * dev_cptr;
    /* dev_bptr: 1st gpu. */
    sptNnzIndex * dev_bptr;
    /* binds_header: 1st cpu, 2nd cpu (ghost pointers) */
    sptBlockIndex ** binds_header = new sptBlockIndex *[nmodes];
    /* dev_binds: 1st gpu, 2nd gpu. */
    sptBlockIndex ** dev_binds;
    /* einds_header: 1st cpu, 2nd cpu (ghost pointers) */
    sptElementIndex ** einds_header = new sptElementIndex *[nmodes];
    /* dev_einds: 1st gpu, 2nd gpu. */
    sptElementIndex ** dev_einds;
    /* dev_values: 1st gpu. */
    sptValue * dev_values;


    /* Copy matrices */
    /* dev_mats_order: 1st gpu. */
    sptIndex * dev_mats_order;
    /* mats_header: 1st cpu, 2nd cpu (ghost pointers) */
    sptValue ** mats_header = new sptValue *[nmodes+1];
    /* lengths: 1st cpu, store the lengths of mats */
    sptIndex * const lengths = new sptIndex[nmodes+1];
    /* dev_mats: 1st gpu, 2nd gpu. */
    sptValue ** dev_mats;
    /* dev_scratch: 1st gpu. */
    sptValue * dev_scratch;
    /* the pointer to dev_mats[nmodes] */
    sptValue *dev_part_prod;  
    sptNnzIndex dev_mem_size = 0;
    sptNnzIndex dev_flops = 2 * nnz * R + (nmodes - 1) * R;


    sptStartTimer(timer);

    /* dev_ndims */
    result = sptCudaDuplicateMemory(&dev_ndims, ndims, nmodes * sizeof (*dev_ndims), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += nmodes * sizeof (*dev_ndims);
    /* dev_cptr */
    result = sptCudaDuplicateMemory(&dev_cptr, hitsr->cptr.data, hitsr->cptr.len * sizeof (*dev_cptr), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += hitsr->cptr.len * sizeof (*dev_cptr);
    /* dev_bptr */
    result = sptCudaDuplicateMemory(&dev_bptr, hitsr->bptr.data, hitsr->bptr.len * sizeof (*dev_bptr), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += hitsr->bptr.len * sizeof (*dev_bptr);
    /* binds_header */
    for(sptIndex m = 0; m < nmodes; ++m) {
        binds_header[m] = hitsr->binds[m].data;
    }
    /* dev_binds */
    result = sptCudaDuplicateMemoryIndirect(&dev_binds, binds_header, nmodes, hitsr->binds[0].len, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += nmodes * hitsr->binds[0].len * sizeof(**dev_binds);
    /* einds_header */
    for(sptIndex m = 0; m < nmodes; ++m) {
        einds_header[m] = hitsr->einds[m].data;
    }
    /* dev_einds */
    result = sptCudaDuplicateMemoryIndirect(&dev_einds, einds_header, nmodes, nnz, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += nmodes * nnz * sizeof(**dev_einds);
    /* dev_values */
    result = sptCudaDuplicateMemory(&dev_values, hitsr->values.data, nnz * sizeof (*dev_values), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += nnz * sizeof (*dev_values);


    /* dev_mats_order */
    result = sptCudaDuplicateMemory(&dev_mats_order, mats_order, nmodes * sizeof (*dev_mats_order), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += nmodes * sizeof (*dev_mats_order);

    /* mats_header and lengths */
    memset(mats[nmodes]->values, 0, mats[mode]->nrows * stride * sizeof(sptValue));
    sptNnzIndex sum_mat_length = 0;
    for(sptIndex m = 0; m < nmodes; ++m) {
        mats_header[m] = mats[m]->values;
        lengths[m] = mats[m]->nrows * stride;
        sum_mat_length += mats[m]->nrows * stride;
    }
    mats_header[nmodes] = mats[nmodes]->values;
    lengths[nmodes] = mats[mode]->nrows * stride;
    sum_mat_length += mats[mode]->nrows * stride;
    /* dev_mats */
    result = sptCudaDuplicateMemoryIndirect(&dev_mats, mats_header, nmodes+1, lengths, hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += sum_mat_length * sizeof(**dev_mats);

    if(nmodes > 4) {
        /* dev_scratch */
        result = hipMalloc((void **) &dev_scratch, nnz * stride * sizeof (*dev_scratch));
        spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
        result = hipMemset(dev_scratch, 0, nnz * stride * sizeof (*dev_scratch));
        spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
        dev_mem_size +=  nnz * stride * sizeof (*dev_scratch);
    }

    sptStopTimer(timer);
    time_h2d = sptElapsedTime(timer);
    gbw_h2d = dev_mem_size / time_h2d /1e9;
    sptPrintElapsedTime(timer, "CUDA HiCOO SpTns MTTKRP H2D");
    printf("[Bandwidth H2D]: %lf GBytes/sec\n", gbw_h2d);

    sptStartTimer(timer);

    /* Loop kernels */
    // for(sptIndex k=0; k<hitsr->kptr.len - 1; ++k) {
    //     sptNnzIndex kptr_begin = hitsr->kptr.data[k];
    //     sptNnzIndex kptr_end = hitsr->kptr.data[k+1];
        sptNnzIndex kptr_begin = hitsr->kptr.data[0];   //useless
        sptNnzIndex kptr_end = hitsr->kptr.data[1];   //useless

        sptAssert( sptMTTKRPKernelHiCOO(
            mode,
            nmodes,
            nnz,
            max_nnzb,
            R,
            stride,
            hitsr->sb_bits,
            hitsr->sc_bits,
            hitsr->bptr.len-1,
            impl_num,
            kptr_begin,
            kptr_end,
            dev_ndims,
            dev_cptr,
            dev_bptr,
            dev_binds,
            dev_einds,
            dev_values,
            dev_mats_order,
            dev_mats) == 0 );

    // }   // End loop kernels
    sptStopTimer(timer);
    time_exe = sptElapsedTime(timer);
    gflops_exe = dev_flops / time_exe / 1e9;
    sptPrintElapsedTime(timer, "CUDA HiCOO SpTns MTTKRP");
    printf("[GFLOPS]: %lf GFlops \n", gflops_exe);

    sptStartTimer(timer);

    dev_mem_size = 0;
    /* Copy back the pointer to dev_mats[nmodes] to the result */
    result = hipMemcpy(&dev_part_prod, dev_mats + nmodes, sizeof dev_part_prod, hipMemcpyDeviceToHost);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += sizeof dev_part_prod;

    result = hipMemcpy(mats[nmodes]->values, dev_part_prod, mats[mode]->nrows * stride * sizeof (*dev_part_prod), hipMemcpyDeviceToHost);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns SpltMTTKRP");
    dev_mem_size += mats[mode]->nrows * stride * sizeof (*dev_part_prod);

    sptStopTimer(timer);
    time_d2h = sptElapsedTime(timer);
    gbw_d2h = dev_mem_size / time_d2h /1e9;
    sptPrintElapsedTime(timer, "CUDA HiCOO SpTns MTTKRP D2H");
    printf("[Bandwidth D2H]: %lf GBytes/sec\n", gbw_d2h);
    sptFreeTimer(timer);


    result = hipFree(dev_ndims);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    result = hipFree(dev_cptr);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    result = hipFree(dev_bptr);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    result = hipFree(dev_binds);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    result = hipFree(dev_einds);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    result = hipFree(dev_values);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");

    result = hipFree(dev_mats_order);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    result = hipFree(dev_mats);
    spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    if(nmodes > 4) {
        result = hipFree(dev_scratch);
        spt_CheckCudaError(result != 0, "CUDA HiCOO SpTns MTTKRP");
    }
    delete[] binds_header;
    delete[] einds_header;
    delete[] mats_header;
    delete[] lengths;

  return 0;
}


