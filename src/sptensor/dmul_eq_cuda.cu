#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include "sptensor.h"

__global__ static void spt_DotMulKernel(size_t nnz, sptValue *Z_val, sptValue *X_val, sptValue *Y_val)
{
    const sptNnzIndex tidx = threadIdx.x;
    const sptNnzIndex i = (sptNnzIndex) (blockIdx.x * blockDim.x + tidx);

    if(i < nnz) {
        Z_val[i] = X_val[i] * Y_val[i];
    }
    __syncthreads();
}



/**
 * CUDA parallelized Element wise multiply two sparse tensors, with exactly the same nonzero
 * distribution.
 * @param[out] Z the result of X*Y, should be uninitialized
 * @param[in]  X the input X
 * @param[in]  Y the input Y
 */
int sptCudaSparseTensorDotMulEq(sptSparseTensor *Z, const sptSparseTensor *X, const sptSparseTensor *Y) {
    int result;
    /* Ensure X and Y are in same shape */
    if(Y->nmodes != X->nmodes) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns DotMul", "shape mismatch");
    }
    for(sptIndex i = 0; i < X->nmodes; ++i) {
        if(Y->ndims[i] != X->ndims[i]) {
            spt_CheckError(SPTERR_SHAPE_MISMATCH, "CUDA SpTns DotMul", "shape mismatch");
        }
    }
    /* Ensure X and Y have exactly the same nonzero distribution */
    if(Y->nnz != X->nnz) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "SpTns DotMul", "nonzero distribution mismatch");
    }
    sptNnzIndex nnz = X->nnz;

    sptCopySparseTensor(Z, X, 1);

    sptValue *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptValue), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    sptValue *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipMemcpy(Y_val, Y->values.data, Y->nnz * sizeof (sptValue), hipMemcpyHostToDevice);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    sptValue *Z_val = NULL;
    result = hipMalloc((void **) &Z_val, X->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipMemset(Z_val, 0, X->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    sptNnzIndex nthreads = 128;
    sptNnzIndex nblocks = (nnz + nthreads -1)/ nthreads;

    sptTimer timer;
    sptNewTimer(&timer, 0);
    sptStartTimer(timer);

    spt_DotMulKernel<<<nblocks, nthreads>>>(nnz, Z_val, X_val, Y_val);
    result = hipDeviceSynchronize();

    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "CUDA  SpTns DotMul");
    sptFreeTimer(timer);

    hipMemcpy(Z->values.data, Z_val, Z->nnz * sizeof (sptValue), hipMemcpyDeviceToHost);

    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipFree(Y_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");
    result = hipFree(Z_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns DotMul");

    /* Check whether elements become zero after adding.
       If so, fill the gap with the [nnz-1]'th element.
    */
    spt_SparseTensorCollectZeros(Z);
    /* Sort the indices */
    sptSparseTensorSortIndex(Z, 1, 1);

    return 0;
}
