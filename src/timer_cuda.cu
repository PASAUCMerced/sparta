/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "error/error.h"

#ifndef PARTI_USE_CUDA

struct sptTagTimer {
    int use_cuda;
    struct timespec start_timespec;
    struct timespec stop_timespec;
    hipEvent_t start_event;
    hipEvent_t stop_event;
};

int sptNewTimer(sptTimer *timer, int use_cuda) {
    *timer = (sptTimer) malloc(sizeof **timer);
    (*timer)->use_cuda = use_cuda;
    if(use_cuda) {
        int result;
        result = hipEventCreate(&(*timer)->start_event);
        spt_CheckCudaError(result, "Timer New");
        result = hipEventCreate(&(*timer)->stop_event);
        spt_CheckCudaError(result, "Timer New");
    }
    return 0;
}

int sptStartTimer(sptTimer timer) {
    int result;
    if(timer->use_cuda) {
        result = hipEventRecord(timer->start_event);
        spt_CheckCudaError(result, "Timer New");
        result = hipEventSynchronize(timer->start_event);
        spt_CheckCudaError(result, "Timer New");
    } else {
        result = clock_gettime(CLOCK_MONOTONIC, &timer->start_timespec);
        spt_CheckOSError(result, "Timer New");
    }
    return 0;
}

int sptStopTimer(sptTimer timer) {
    int result;
    if(timer->use_cuda) {
        result = hipEventRecord(timer->stop_event);
        spt_CheckCudaError(result, "Timer New");
        result = hipEventSynchronize(timer->stop_event);
        spt_CheckCudaError(result, "Timer New");
    } else {
        result = clock_gettime(CLOCK_MONOTONIC, &timer->stop_timespec);
        spt_CheckOSError(result, "Timer New");
    }
    return 0;
}

double sptElapsedTime(const sptTimer timer) {
    if(timer->use_cuda) {
        float elapsed;
        if(hipEventElapsedTime(&elapsed, timer->start_event, timer->stop_event) != 0) {
            return NAN;
        }
        return elapsed * 1e-3;
    } else {
        return timer->stop_timespec.tv_sec - timer->start_timespec.tv_sec
            + (timer->stop_timespec.tv_nsec - timer->start_timespec.tv_nsec) * 1e-9;
    }
}

double sptPrintElapsedTime(const sptTimer timer, const char *name) {
    double elapsed_time = sptElapsedTime(timer);
    fprintf(stdout, "[%s]: %.9lf s\n", name, elapsed_time);
    return elapsed_time;
}


double sptPrintAverageElapsedTime(const sptTimer timer, const int niters, const char *name) {
    double elapsed_time = sptElapsedTime(timer) / niters;
    fprintf(stdout, "[%s]: %.9lf s\n", name, elapsed_time);
    return elapsed_time;
}

int sptFreeTimer(sptTimer timer) {
    if(timer->use_cuda) {
        int result;
        result = hipEventDestroy(timer->start_event);
        spt_CheckCudaError(result, "Timer New");
        result = hipEventDestroy(timer->stop_event);
        spt_CheckCudaError(result, "Timer New");
    }
    free(timer);
    return 0;
}

#endif